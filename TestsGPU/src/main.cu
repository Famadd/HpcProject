#include "../includes/config.h"
#include "mat.h"

int main(int argc, char **argv) {

    int taille = 0;

    std::cout << "Choissisez la taille (nombres de colonnes ou lignes) des matrices à calculer" << std::endl;
    do{
        std::cout << "La taille doit etre comprise entre 0 et 15000" << std::endl;
        std::cin >> taille;
    } while (taille < 0 || taille > 15000); 

    size_t size = taille*taille*sizeof(float);

    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((taille+15)/16, (taille+15)/16);


    Mat<float> h_A(taille,taille,0.0f);
    Mat<float> h_B(taille,taille,0.0f);
    Mat<float> h_C(taille,taille,0.0f);

    h_A.fillRandNumber(0,2000.0f);
    h_B.fillRandNumber(0,2000.0f);
    h_C.fillRandNumber(0,2000.0f);

    float* d_A;
    float* d_B;
    float* d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(&d_A, h_A.getDataPtr(), size, hipMemcpyHostToDevice);
    hipMemcpy(&d_B, h_B.getDataPtr(), size, hipMemcpyHostToDevice);
    hipMemcpy(&d_C, h_C.getDataPtr(), size, hipMemcpyHostToDevice);

    matrixProductTimeCalculus<float>(h_A, h_B, h_C,taille);


    matrixAdditionTimeCalculus<float>(h_A, h_B, h_C,taille);


    hadamardProductTimeCalculus(numBlocks, threadsPerBlock, d_A, d_B, d_C, taille, taille);

    hipDeviceSynchronize();

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
